
#include <hip/hip_runtime.h>
#include <stdint.h>

__device__ void dump_bucket(uint64_t *buckets,
                            uint32_t num_ranges, uint32_t tile_size,
                            uint32_t src_i, uint32_t src_j,
                            uint32_t dest_i, uint32_t dest_j) {
    // Element-wise sum for each in 0 -> num_ranges.
    for (uint32_t k = 0; k < num_ranges; k++) {
        uint32_t src_index = (tile_size * tile_size * k) +
                             (tile_size * src_i) + src_j;
        uint32_t dest_index = (tile_size * tile_size * k) +
                              (tile_size * dest_i) + dest_j;
        buckets[dest_index] += buckets[src_index];
    }
}

__global__ void reduction(uint64_t *buckets, uint32_t num_ranges,
                          uint32_t tile_size, uint32_t chunk_size) {
    // Calculate <i, j> coords within the tile.
    uint32_t i = blockIdx.x; // row
    uint32_t j = threadIdx.x * chunk_size; // column

    // Each chunk leader reduces its chunk.
    for (uint32_t k = 1; k < chunk_size; k++) {
        dump_bucket(buckets, num_ranges, tile_size, i, j + k, i, j);
    }

    // Wait for all the threads in this row to finish.
    __syncthreads();

    // Reduce each chunk leader into the zeroth element of the row.
    if (j == 0) {
        for (uint32_t k = 1; k < blockDim.x; k++) {
            dump_bucket(buckets, num_ranges, tile_size, i, k * chunk_size, i, 0);
        }
    }
}

__global__ void pearson(uint64_t *buckets,
                        float *ranges, uint32_t num_ranges,
                        float *A, float *B,
                        uint32_t tile_size, uint32_t s, uint32_t t,
                        uint32_t n, uint32_t m, uint32_t p) {
    // Calculate relative <i, j> coords within this tile.
    uint32_t i = blockIdx.y * blockDim.y + threadIdx.y; // row
    uint32_t j = blockIdx.x * blockDim.x + threadIdx.x; // column

    // Calculate the offsets based on the tile number.
    uint32_t i_offset = s * tile_size;
    uint32_t j_offset = t * tile_size;

    // Calculate the absolute <i, j> coords within the matrix.
    uint32_t i_abs = i_offset + i;
    uint32_t j_abs = j_offset + j;

    // Only compute values inside the bounds of the matrix.
    if (i_abs >= n || j_abs >= m) {
        return;
    }

    // Initialize accumulators and the result.
    float sum_x, sum_y, sum_x2, sum_y2, sum_xy, coeff;
    sum_x = sum_y = sum_x2 = sum_y2 = sum_xy = coeff = 0.0f;

    // Compute the sums.
    for (uint32_t k = 0; k < p; k++) {
        float x = A[i * p + k];
        float y = B[j * p + k];

        sum_x += x;
        sum_y += y;
        sum_x2 += x * x;
        sum_y2 += y * y;
        sum_xy += x * y;
    }

    // Compute the Pearson coefficient using the "sometimes numerically
    // unstable" method because it's way more computationally efficient.
    coeff = (p * sum_xy - sum_x * sum_y) /
            sqrtf((p * sum_x2 - sum_x * sum_x) * (p * sum_y2 - sum_y * sum_y));

    // Dump it in the appropriate bucket.
    for (uint32_t k = 0; k < num_ranges; k++) {
        float low = ranges[2 * k + 0];
        float high = ranges[2 * k + 1];
        if (coeff >= low && coeff < high) {
            uint32_t index = (tile_size * tile_size * k) +
                             (tile_size * i) + j;
            buckets[index] += 1;
        }
    }
}
